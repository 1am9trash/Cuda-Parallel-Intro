#include <hip/hip_runtime.h>
#include "gpu_utils.cuh"


GPUTimer::GPUTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

GPUTimer::~GPUTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void GPUTimer::start_timer() {
    hipEventRecord(start, 0);
}

void GPUTimer::stop_timer() {
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
}

float GPUTimer::get_elaspsed_time_in_ms() {
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    return ms;
}